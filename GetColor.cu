#include "hip/hip_runtime.h"
﻿//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>


extern "C" {
	__device__
    inline float getDotProduct(float3 a, float3 b)
	{
		return a.x * b.x + a.y * b.y + a.z * b.z;
	}
	
	__device__ 
	float3 getNormalizedVec(const float3 v)
	{
		float invLen = 1.0f / sqrtf(getDotProduct(v, v));
		return make_float3(v.x * invLen, v.y * invLen, v.z * invLen);
	}

	__device__
	float3 operator*(const float3 &a, const float &b) {
		return make_float3(a.x * b, a.y * b, a.z * b);
	}
	
	__device__
	float3 operator-(const float3 &a, const float3 &b) {
		return make_float3(a.x-b.x, a.y-b.y, a.z-b.y);
	}

	
	struct Test {
		float X;
		float Y;
		float Z;
	};
	
	struct PointStruct {
		float X;
		float Y;
		float Z;
	};
	
	struct ColorStruct {
		int R;
		int G;
		int B;
	};
	
	struct PixelStruct {
		ColorStruct color;
		float Z;
		PointStruct point;
	};

	__device__
	int rgbToInt(int r, int g, int b) {
		return 0x00000000 | r << 16 | g << 8 | b | 255 << 24;
	}

	// Device code
	__global__ void VecAdd(
		struct PixelStruct* pixels,
		
		struct ColorStruct lightColor, 
		struct PointStruct lightV, 
		
		float kD,
		float kS,
		int m,
		
		int n, 
		int *result
	) {
		int index = blockDim.x * blockIdx.x + threadIdx.x;
	
		if (index >= n) return;
	
		int y = index / 500;
		int x = index - y * 500;
		
		struct PixelStruct pix = pixels[index];
	
		if (pix.Z == 0) {
			result[index] = rgbToInt(127, 127, 255);
			return;
		}
	
		float3 L = getNormalizedVec(make_float3(lightV.X - (float)x, lightV.Y - (float)y, lightV.Z - pix.Z));
		float3 N = getNormalizedVec(make_float3(pix.point.X, pix.point.Y, pix.point.Z));

		float nCosL = getDotProduct(N, L);
		float3 R = getNormalizedVec(N * (2.0f * nCosL) - L);
		float vCosR = R.z;
		
		float first = kD * max(.0f, nCosL);
		float second = kS * pow(vCosR, (float)m);
		
		int r, g, b;
		
		// Red
		float iLiOR = ((float)lightColor.R / 255.0f) * (float)pix.color.R;
		int colorR = (int)(first * iLiOR + second * iLiOR);
		r = max(0, min(255, colorR));
		
		// Green
		float iLiOG = ((float)lightColor.G / 255.0f) * (float)pix.color.G;
		int colorG = (int)(first * iLiOG + second * iLiOG);
		g = max(0, min(255, colorG));
		
		// Blue
		float iLiOB = ((float)lightColor.B / 255.0f) * (float)pix.color.B;
		int colorB = (int)(first * iLiOB + second * iLiOB);
		b = max(0, min(255, colorB));
		
		result[index] = rgbToInt(r, g, b);
	}
	
}

int main() { 
	return 0; 
}